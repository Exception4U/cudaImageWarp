#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <sift3d/imtypes.h>

#include "cudaImageWarp.h" // Need this to get C linkage on exported functions

#define DIVC(x, y)  (((x) + (y) + 1) / (y)) // Divide integers and ceil
#define AFFINE_WARP(x, y, z, f4) /* Warp using a float4 */ \
        (x * f4.x + y * f4.y + z * f4.z + f4.w)

/********************/
/* CUDA ERROR CHECK */
/********************/
typedef unsigned int  uint;

texture<float, 3, hipReadModeElementType> tex;  // 3D texture

/************************************************/
/* TEXTURE-BASED TRILINEAR INTERPOLATION KERNEL */
/************************************************/
__global__ void
warp(float *d_output, const uint nx, const uint ny, const uint nz, 
        const float4 xWarp, const float4 yWarp, const float4 zWarp)
{
    const uint x = blockIdx.x * blockDim.x + threadIdx.x;
    const uint y = blockIdx.y * blockDim.y + threadIdx.y;
    const uint z = blockIdx.z * blockDim.z + threadIdx.z;

    // Check boundaries
    if (x >= nx || y >= ny || z >= nz)
        return;

    const uint y_stride = nx;
    const uint z_stride = nx * ny;

    // Read from the 3D texture
    const float xs = AFFINE_WARP(x, y, z, xWarp);
    const float ys = AFFINE_WARP(x, y, z, yWarp);
    const float zs = AFFINE_WARP(x, y, z, zWarp);
    const float voxel = tex3D(tex, xs, ys, zs);

    // Write the output
    const uint idx = z * z_stride + y * y_stride + x;
    d_output[idx] = voxel;
}

/* Warp an image in-place.  Params is an array of 12 floats, in row-major 
 * order. */
int cuda_image_warp(Image *const im, const float *const params) {

    // Convert the input
    const float4 xWarp = {params[0], params[1], params[2], params[3]};
    const float4 yWarp = {params[4], params[5], params[6], params[7]};
    const float4 zWarp = {params[8], params[9], params[10], params[11]}; 

    // Intermediates
    float *d_output = NULL;

#define CLEANUP { \
    if (d_output != NULL) \
        hipFree(d_output); \
} 

#define gpuAssert(code, file, line) { \
    if (code != hipSuccess) { \
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), \
                file, line); \
        CLEANUP \
        return -1; \
       }  \
} \

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

    // --- Allocate device memory for output
    const size_t im_mem_size = im->size * sizeof(float);
    gpuErrchk(hipMalloc((void**)&d_output, im_mem_size));

    // --- Create 3D array
    const hipExtent volumeSize = make_hipExtent(im->nx, im->ny, im->nz);

    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
    hipArray *d_inputArray = 0;
    gpuErrchk(hipMalloc3DArray(&d_inputArray, &channelDesc, volumeSize));

    // --- Copy data to 3D array (host to device)
    hipMemcpy3DParms copyParams = {0};
    copyParams.srcPtr   = make_hipPitchedPtr(im->data, 
        im->nx * sizeof(float), im->nx, im->ny);
    copyParams.dstArray = d_inputArray;
    copyParams.extent   = volumeSize;
    copyParams.kind     = hipMemcpyHostToDevice;
    gpuErrchk(hipMemcpy3D(&copyParams));

    // --- Set texture parameters
    tex.normalized = false; // access with un-normalized texture coordinates
    tex.filterMode = hipFilterModeLinear; // linear interpolation
    tex.addressMode[0] = hipAddressModeBorder; // wrap texture coordinates
    tex.addressMode[1] = hipAddressModeBorder;
    tex.addressMode[2] = hipAddressModeBorder;

    // --- Bind array to 3D texture
    gpuErrchk(hipBindTextureToArray(tex, d_inputArray, channelDesc));

    // --- Launch the interpolation kernel
    const dim3 blockSize(16, 16, 1);
    const dim3 gridSize(DIVC(im->nx, blockSize.x), DIVC(im->ny, blockSize.y),
            DIVC(im->nz, blockSize.z));
    warp<<<gridSize, blockSize>>>(d_output, im->nx, im->ny, im->nz, xWarp,
        yWarp, zWarp);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());

    // --- Copy the interpolated data to the host, in-place
    gpuErrchk(hipMemcpy(im->data,d_output,im_mem_size,hipMemcpyDeviceToHost));

    CLEANUP
    return 0;

#undef CLEANUP
}
