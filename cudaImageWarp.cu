#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include "cudaImageWarp.h" // Need this to get C linkage on exported functions

#define DIVC(x, y)  (((x) + (y) + 1) / (y)) // Divide positive integers and ceil

/* Global data */
typedef unsigned int  uint;

/* CUDA device function which does the affine warping */
__device__ float affine_warp(const uint x, const uint y, const uint z, 
	const float4 warp) {
        return x * warp.x + y * warp.y + z * warp.z + warp.w;
}

/* Sample from a texture after affine warping */
__device__ float sample_affine(hipTextureObject_t tex,
    float *const output, 
    const uint x, const uint y, const uint z,
    const float4 xWarp, const float4 yWarp, const float4 zWarp) {

    const float xs = affine_warp(x, y, z, xWarp);
    const float ys = affine_warp(x, y, z, yWarp);
    const float zs = affine_warp(x, y, z, zWarp);

    return tex3D<float>(tex, xs, ys, zs);
}

/* Deice function to perform image post-processing */
__device__ float postprocess(const float in, hiprandState_t *state, 
	const float std, const float window_min, const float window_max) {

    // Generate white Gaussian noise, if std > 0
    const float noise = std > 0 ? hiprand_normal(state) * std : 0.0f;

    // Add the noise
    float out = in + noise;

    // Clamp using the window bounds
    out = max(out, window_min);
    out = min(out, window_max);

    // Normalize to [0,1] using the window
    const float window_width = window_max - window_min;
    return isfinite(window_width) ? (out - window_min) / window_width : out;
}

/* Image warping kernel */
__global__ void
warp(hipTextureObject_t tex, float *const output, hiprandState_t *const states,
    const float std, const uint nx, const uint ny, const uint nz, 
    const float window_min, const float window_max,
    const float4 xWarp, const float4 yWarp, const float4 zWarp)
{

#define CUDA_SET_DIMS \
    const uint x = blockIdx.x * blockDim.x + threadIdx.x; \
    const uint y = blockIdx.y * blockDim.y + threadIdx.y; \
    const uint z = blockIdx.z * blockDim.z + threadIdx.z; \
    \
    /* Check boundaries */ \
    if (x >= nx || y >= ny || z >= nz) \
	return; \
    \
    const uint y_stride = nx; \
    const uint z_stride = nx * ny; \
    const uint idx = z * z_stride + y * y_stride + x; 

    CUDA_SET_DIMS

    // Read from the 3D texture and postprocess
    const float in = sample_affine(tex, output, x, y, z, xWarp, yWarp, zWarp);
    output[idx] =  postprocess(in, states + idx, std, window_min, window_max);
}

/* Initialize an RNG for each thread. This uses a different seed for each
 * generator. This is much faster than using separate sequences with the same
 * seed, but we are not guaranteed independence between generators. */
__global__ void initRand(const int seed, hiprandState_t *const states,
    const uint nx, const uint ny, const uint nz) {
    CUDA_SET_DIMS // See warp()
    hiprand_init(seed + idx, 0, 0, states + idx);
}

/* Warp an image in-place.  
* Parameters:
*  input - an array of nxi * nyi * nzi floats, strided in (x,y,z) order
*  nxi, nyi, nzi - the input image dimensions
*  output - an array of nxo * nyo * nzo floats, strided in (x,y,z) order
*  nxo, nyo, nzo - the output image dimensions
*  filter_mode - use 0 for nearest neighbor, 1 for linear
*  params - an array of 12 floats, in row-major order
*  std - standard deviation for additive white Gaussian noise. Disables noise if
*	std <= 0.
*  window_min - the minimum value for the window. Use -INFINITY to do nothing.
*  window_max - the maximum value for the window. Use INFINITY to do nothing.
*
* Returns 0 on success, nonzero otherwise. */
int cuda_image_warp(const float *const input, 
    const int nxi, const int nyi, const int nzi, 
    float *const output,
    const int nxo, const int nyo, const int nzo, 
    const int filter_mode, const float *const params, const float std,
    const float window_min, const float window_max) {

    // Convert the input to CUDA datatypes
    const float4 xWarp = {params[0], params[1], params[2], params[3]};
    const float4 yWarp = {params[4], params[5], params[6], params[7]};
    const float4 zWarp = {params[8], params[9], params[10], params[11]}; 

    // Intermediates
    float *d_output = NULL;
    hiprandState_t *d_states = NULL;
    hipArray *d_input = NULL;

#define gpuAssert(code, file, line) { \
if (code != hipSuccess) { \
    fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), \
	    file, line); \
    if (d_output != NULL) \
	hipFree(d_output); \
    if (d_states != NULL) \
	hipFree(d_states); \
    if (d_input != NULL) \
	hipFreeArray(d_input); \
    return -1; \
}  \
} \

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

#define CLEANUP { \
if (d_output != NULL) { \
    hipFree(d_output); \
    gpuErrchk(hipPeekAtLastError()); \
} \
if (d_states != NULL) { \
    hipFree(d_states); \
    gpuErrchk(hipPeekAtLastError()); \
} \
if (d_input != NULL) { \
    hipFreeArray(d_input); \
    gpuErrchk(hipPeekAtLastError()); \
} \
} 

    // --- Allocate device memory for output
    const size_t num_voxels = nxo * nyo * nzo;
    const size_t out_size = num_voxels * sizeof(float);
    gpuErrchk(hipMalloc((void**) &d_output, out_size));
    gpuErrchk(hipMalloc((void**) &d_states, 
	num_voxels * sizeof(hiprandState_t)));

    // --- Create 3D array
    const hipExtent inVolumeSize = make_hipExtent(nxi, nyi, nzi);

    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
    gpuErrchk(hipMalloc3DArray(&d_input, &channelDesc, inVolumeSize));

    // --- Copy the input data to a 3D array (host to device)
    hipMemcpy3DParms copyParams = {0};
    copyParams.srcPtr   = make_hipPitchedPtr((void *) input,
	nxi * sizeof(float), nxi, nyi);
    copyParams.dstArray = d_input;
    copyParams.extent   = inVolumeSize;
    copyParams.kind     = hipMemcpyHostToDevice;
    gpuErrchk(hipMemcpy3D(&copyParams));

    // --- Create the texture object
    hipTextureObject_t tex;
    hipResourceDesc    texRes;
    memset(&texRes, 0, sizeof(hipResourceDesc));
    texRes.resType = hipResourceTypeArray;
    texRes.res.array.array  = d_input;
    hipTextureDesc texDescr;
    memset(&texDescr, 0, sizeof(hipTextureDesc));
    texDescr.normalizedCoords = false;
    texDescr.addressMode[0] = hipAddressModeBorder;
    texDescr.addressMode[1] = hipAddressModeBorder;
    texDescr.addressMode[2] = hipAddressModeBorder;
    texDescr.readMode = hipReadModeElementType;
    switch (filter_mode) {
	case 0:
	    // Nearest neighbor interpolation
	    texDescr.filterMode = hipFilterModePoint;
	    break;
	case 1:
	    // Linear interpolation
	    texDescr.filterMode = hipFilterModeLinear;
	    break;
	default:
	    fprintf(stderr, "Unrecognized filter_mode: %d \n", filter_mode);
            CLEANUP
	    return -1;
    }
    gpuErrchk(hipCreateTextureObject(&tex, &texRes, &texDescr, NULL));

    // Configure the block and grid sizes
    const dim3 blockSize(16, 16, 1);
    const dim3 gridSize(DIVC(nxo, blockSize.x), DIVC(nyo, blockSize.y),
	DIVC(nzo, blockSize.z));

    // Initialize the random number generators
    //TODO we could keep track of the last image size globally, only
    // calling this kernel if that number changes
    if (std > 0.0f) { 
	// Get the random seed from the time
	const time_t seed = clock();

	// Initialize one RNG per thread
	initRand<<<gridSize, blockSize>>>(seed, d_states, nxo, nyo, nzo);
    }

    // Perform image warping and augmentation
    warp<<<gridSize, blockSize>>>(tex, d_output, d_states, std, nxo, nyo, nzo, 
	window_min, window_max, xWarp, yWarp, zWarp);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());

    // --- Copy the output data to the host
    gpuErrchk(hipMemcpy(output,d_output,out_size,hipMemcpyDeviceToHost));

    // Destroy the texture object
    hipDestroyTextureObject(tex);  

    CLEANUP
    return 0;

#undef CLEANUP
}

